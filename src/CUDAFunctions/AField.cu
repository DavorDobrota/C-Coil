#include "hip/hip_runtime.h"
#include "hardware_acceleration.h"

#include "CUDAConstants.h"
#include "Timing.h"
#include "CUDAErrorCheck.h"
#include "CoilData.h"

#include <cstdio>


__global__
void calculateA(long long numOps, CoilData coil,
                const TYPE *xPosArr, const TYPE *yPosArr, const TYPE *zPosArr,
                TYPE *xResArr, TYPE *yResArr, TYPE *zResArr)
{
    unsigned int index = threadIdx.x;
    long long global_index = blockIdx.x * blockDim.x + index;

    if(global_index >= numOps)
        return;

    TYPE x1 = xPosArr[global_index];
    TYPE y1 = yPosArr[global_index];
    TYPE z1 = zPosArr[global_index];

    x1 -= coil.positionVector[0];
    y1 -= coil.positionVector[1];
    z1 -= coil.positionVector[2];

    TYPE x = x1 * coil.invTransformArray[0] + y1 * coil.invTransformArray[1] + z1 * coil.invTransformArray[2];
    TYPE y = x1 * coil.invTransformArray[3] + y1 * coil.invTransformArray[4] + z1 * coil.invTransformArray[5];
    TYPE z = x1 * coil.invTransformArray[6] + y1 * coil.invTransformArray[7] + z1 * coil.invTransformArray[8];

    TYPE zCoord = z;
    TYPE rCoord = sqrt(x * x + y * y);
    TYPE phiCord = atan2(y, x);

    TYPE potential = 0.0f;
    TYPE constant = coil.constFactor;

    TYPE topEdge = zCoord + 0.5f * coil.length;
    TYPE bottomEdge = zCoord - 0.5f * coil.length;

    for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
    {
        TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.positionArray[incT]);

        for (int incF = 0; incF < coil.angularIncrements; ++incF)
        {
            TYPE cosinePhi = coil.cosPrecomputeArray[incF];

            TYPE tempConstC = rsqrt(incrementPositionT * incrementPositionT + rCoord * rCoord -
                                     2.0f * incrementPositionT * rCoord * cosinePhi);

            TYPE tempConstD1 = topEdge * tempConstC;
            TYPE tempConstD2 = bottomEdge * tempConstC;

            TYPE tempConstE1 = sqrt(tempConstD1 * tempConstD1 + 1.0f);
            TYPE tempConstE2 = sqrt(tempConstD2 * tempConstD2 + 1.0f);

            TYPE tempConstF = log((tempConstE1 + tempConstD1) / (tempConstE2 + tempConstD2));

            potential += constant *
                    coil.weightArray[incT] * coil.weightArray[incF] *
                    incrementPositionT * cosinePhi * tempConstF;
        }
    }
    TYPE xPot = (-1.f) * sin(phiCord) * potential;
    TYPE yPot = potential * cos(phiCord);
    TYPE zPot = 0.f;

    TYPE xRes = xPot * coil.transformArray[0] + yPot * coil.transformArray[1] + zPot * coil.transformArray[2];
    TYPE yRes = xPot * coil.transformArray[3] + yPot * coil.transformArray[4] + zPot * coil.transformArray[5];
    TYPE zRes = xPot * coil.transformArray[6] + yPot * coil.transformArray[7] + zPot * coil.transformArray[8];

    xResArr[global_index] = xRes;
    yResArr[global_index] = yRes;
    zResArr[global_index] = zRes;
}
	
namespace 
{
    long long g_last_num_ops = 0;

    TYPE *g_xPosArr = nullptr;
    TYPE *g_yPosArr = nullptr;
    TYPE *g_zPosArr = nullptr;

    TYPE *g_xResArr = nullptr;
    TYPE *g_yResArr = nullptr;
    TYPE *g_zResArr = nullptr;
    
    #if DEBUG_TIMINGS
        double g_duration;
    #endif
}

void resourceCleanupA()
{
	gpuErrchk(hipFree(g_xPosArr));
    gpuErrchk(hipFree(g_yPosArr));
    gpuErrchk(hipFree(g_zPosArr));

    gpuErrchk(hipFree(g_xResArr));
    gpuErrchk(hipFree(g_yResArr));
    gpuErrchk(hipFree(g_zResArr));

    g_xPosArr = nullptr;
    g_yPosArr = nullptr;
    g_zPosArr = nullptr;

    g_xResArr = nullptr;
    g_yResArr = nullptr;
    g_zResArr = nullptr;
}

void resourceStartupA(long long numOps)
{
    resourceCleanupA();
    
	gpuErrchk(hipMalloc(&g_xPosArr, numOps * sizeof(TYPE)));
    gpuErrchk(hipMalloc(&g_yPosArr, numOps * sizeof(TYPE)));
    gpuErrchk(hipMalloc(&g_zPosArr, numOps * sizeof(TYPE)));

    gpuErrchk(hipMalloc(&g_xResArr, numOps * sizeof(TYPE)));
    gpuErrchk(hipMalloc(&g_yResArr, numOps * sizeof(TYPE)));
    gpuErrchk(hipMalloc(&g_zResArr, numOps * sizeof(TYPE)));
}


void Calculate_hardware_accelerated_a (long long numOps, CoilData coil,
                                       const TYPE *xPosArr, const TYPE *yPosArr, const TYPE *zPosArr,
                                       TYPE *xResArr, TYPE *yResArr, TYPE *zResArr)
{
    #if DEBUG_TIMINGS
        recordStartPoint();
    #endif

    long long blocks = ceil(double(numOps) / NTHREADS);

    if (numOps > g_last_num_ops)
    {
        resourceStartupA(numOps);
        g_last_num_ops = numOps;
    }
    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tResource startup:         %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    gpuErrchk(hipMemcpy(g_xPosArr, xPosArr, numOps * sizeof(TYPE), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(g_yPosArr, yPosArr, numOps * sizeof(TYPE), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(g_zPosArr, zPosArr, numOps * sizeof(TYPE), hipMemcpyHostToDevice));

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tMemory initialization:    %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    calculateA<<<blocks, NTHREADS>>>(numOps, coil,
                                     g_xPosArr, g_yPosArr, g_zPosArr,
                                     g_xResArr, g_yResArr, g_zResArr);
	gpuErrchk(hipDeviceSynchronize());

	#if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\t\tCalculations:     %.9g s\n", g_duration);
        printf("\t\tEstimated TFLOPS: %.2f\n",
               1e-12 * double(100 * numOps * coil.thicknessIncrements * coil.angularIncrements) / g_duration);

        recordStartPoint();
    #endif

	if(xResArr != nullptr)
    {
        gpuErrchk(hipMemcpy(xResArr, g_xResArr, numOps * sizeof(TYPE), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(yResArr, g_yResArr, numOps * sizeof(TYPE), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(zResArr, g_zResArr, numOps * sizeof(TYPE), hipMemcpyDeviceToHost));
    }

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tWriting to output array:  %.9g s\n\n", g_duration);
    #endif

	#if DEBUG_TIMINGS
        printf("\tDevice buffer size:       %.3lf MB\n", (6.0 * double(numOps * sizeof(TYPE)) / 1.0e6));
        printf("\tTotal blocks:             %lli\n", blocks);
        printf("\tThreads per calculation:  %i\n", NTHREADS);
        printf("\tPrecision:                %dx%d\n", coil.thicknessIncrements, coil.angularIncrements);
        printf("\tTotal calculations:       %lli\n", numOps);
        printf("\tTotal MegaIncrements:     %.f\n", 1e-6 * double(numOps * coil.thicknessIncrements * coil.angularIncrements));
    #endif
}
