#include "hip/hip_runtime.h"
#include "hardware_acceleration.h"

#include "CUDAConstants.h"
#include "Timing.h"
#include "CUDAErrorCheck.h"
#include "CoilData.h"

#include <cstdio>


__global__
void calculateFieldSlow(long long numOps, CoilData coil, const DataVector *posArr, DataVector *resArr)
{
    unsigned int index = threadIdx.x;
    long long global_index = blockIdx.x * blockDim.x + index;

    if(global_index >= numOps)
        return;

    TYPE x1 = posArr[global_index].x;
    TYPE y1 = posArr[global_index].y;
    TYPE z1 = posArr[global_index].z;

    x1 -= coil.positionVector[0];
    y1 -= coil.positionVector[1];
    z1 -= coil.positionVector[2];

    TYPE x = x1 * coil.invTransformArray[0] + y1 * coil.invTransformArray[1] + z1 * coil.invTransformArray[2];
    TYPE y = x1 * coil.invTransformArray[3] + y1 * coil.invTransformArray[4] + z1 * coil.invTransformArray[5];
    TYPE z = x1 * coil.invTransformArray[6] + y1 * coil.invTransformArray[7] + z1 * coil.invTransformArray[8];

    TYPE zCoord = z;
    TYPE rCoord = sqrt(x * x + y * y);
    TYPE phiCord = atan2(y, x);

    TYPE fieldH = 0.0f;
    TYPE fieldZ = 0.0f;

    for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
    {
        TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

        TYPE tempConstA = incrementPositionT * incrementPositionT;
        TYPE tempConstB = incrementPositionT * rCoord;
        TYPE tempConstC = tempConstA + rCoord * rCoord + zCoord * zCoord;
        TYPE tempConstD = incrementPositionT * zCoord;

        for (int incF = 0; incF < coil.angularIncrements; ++incF)
        {
            TYPE cosinePhi = coil.cosPrecomputeArray[incF];

            TYPE tempConstE = tempConstC - 2.0f * tempConstB * cosinePhi;
            TYPE tempConstF = tempConstE * sqrt(tempConstE);
            TYPE tempConstG = coil.constFactor * coil.thicknessWeightArray[incT] * coil.angularWeightArray[incF] / tempConstF;

            fieldH += tempConstG * (tempConstD * cosinePhi);
            fieldZ += tempConstG * (tempConstA - tempConstB * cosinePhi);
        }
    }

    TYPE xField = fieldH * cos(phiCord);
    TYPE yField = fieldH * sin(phiCord);
    TYPE zField = fieldZ;

    TYPE xRes = xField * coil.transformArray[0] + yField * coil.transformArray[1] + zField * coil.transformArray[2];
    TYPE yRes = xField * coil.transformArray[3] + yField * coil.transformArray[4] + zField * coil.transformArray[5];
    TYPE zRes = xField * coil.transformArray[6] + yField * coil.transformArray[7] + zField * coil.transformArray[8];

    resArr[global_index].x += xRes;
    resArr[global_index].y += yRes;
    resArr[global_index].z += zRes;
}

__global__
void calculateFieldFast(long long numOps, CoilData coil, const DataVector *posArr, DataVector *resArr)
{
    unsigned int index = threadIdx.x;
    long long global_index = blockIdx.x * blockDim.x + index;

    if(global_index >= numOps)
        return;

    TYPE x1 = posArr[global_index].x;
    TYPE y1 = posArr[global_index].y;
    TYPE z1 = posArr[global_index].z;

    x1 -= coil.positionVector[0];
    y1 -= coil.positionVector[1];
    z1 -= coil.positionVector[2];

    TYPE x = x1 * coil.invTransformArray[0] + y1 * coil.invTransformArray[1] + z1 * coil.invTransformArray[2];
    TYPE y = x1 * coil.invTransformArray[3] + y1 * coil.invTransformArray[4] + z1 * coil.invTransformArray[5];
    TYPE z = x1 * coil.invTransformArray[6] + y1 * coil.invTransformArray[7] + z1 * coil.invTransformArray[8];

    TYPE zCoord = z;
    TYPE rCoord = sqrt(x * x + y * y);
    TYPE phiCord = atan2(y, x);

    TYPE fieldH = 0.0f;
    TYPE fieldZ = 0.0f;

    TYPE topEdge = zCoord + 0.5f * coil.length;
    TYPE bottomEdge = zCoord - 0.5f * coil.length;

    for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
    {
        TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

        TYPE tempConstA = incrementPositionT * incrementPositionT;
        TYPE tempConstB = 2.0f * incrementPositionT * rCoord;
        TYPE tempConstC = tempConstA + rCoord * rCoord;

        TYPE tempConstD1 = topEdge * topEdge + tempConstC;
        TYPE tempConstD2 = bottomEdge * bottomEdge + tempConstC;

        for (int incF = 0; incF < coil.angularIncrements; ++incF)
        {
            TYPE cosinePhi = coil.cosPrecomputeArray[incF];

            TYPE tempConstE = tempConstB * cosinePhi;

            TYPE tempConstF1 = rsqrt(tempConstD1 - tempConstE);
            TYPE tempConstF2 = rsqrt(tempConstD2 - tempConstE);

            TYPE tempConstG = coil.constFactor * coil.thicknessWeightArray[incT] * coil.angularWeightArray[incF];

            fieldH += tempConstG * incrementPositionT * cosinePhi * (tempConstF2 - tempConstF1);
            fieldZ += tempConstG *
                    ((tempConstA - 0.5f * tempConstE) / (tempConstC - tempConstE)) *
                    (topEdge * tempConstF1 - bottomEdge * tempConstF2);
        }
    }

    TYPE xField = fieldH * cos(phiCord);
    TYPE yField = fieldH * sin(phiCord);
    TYPE zField = fieldZ;

    TYPE xRes = xField * coil.transformArray[0] + yField * coil.transformArray[1] + zField * coil.transformArray[2];
    TYPE yRes = xField * coil.transformArray[3] + yField * coil.transformArray[4] + zField * coil.transformArray[5];
    TYPE zRes = xField * coil.transformArray[6] + yField * coil.transformArray[7] + zField * coil.transformArray[8];

    resArr[global_index].x += xRes;
    resArr[global_index].y += yRes;
    resArr[global_index].z += zRes;
}

namespace
{
    long long g_last_num_ops = 0;

    DataVector *g_posArr = nullptr;
    DataVector *g_resArr = nullptr;

#if DEBUG_TIMINGS
    double g_duration;
#endif
}

void resourceCleanupB()
{
    gpuErrchk(hipFree(g_posArr));
    gpuErrchk(hipFree(g_resArr));

    g_posArr = nullptr;
    g_resArr = nullptr;
}

void resourceStartupB(long long numOps)
{
    resourceCleanupB();

    gpuErrchk(hipMalloc(&g_posArr, numOps * sizeof(DataVector)));
    gpuErrchk(hipMalloc(&g_resArr, numOps * sizeof(DataVector)));
}


void Calculate_hardware_accelerated_b(long long numOps, CoilData coil,
                                      const DataVector *posArr,
                                      DataVector *resArr)
{
    #if DEBUG_TIMINGS
        recordStartPoint();
        recordStartPoint();
    #endif

    long long blocks = ceil(double(numOps) / NTHREADS);

    if (numOps > g_last_num_ops)
    {
        resourceStartupB(numOps);
        g_last_num_ops = numOps;
    }
    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tResource startup:         %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    gpuErrchk(hipMemcpy(g_posArr, posArr, numOps * sizeof(DataVector), hipMemcpyHostToDevice));

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
            printf("\tMemory initialization:    %.9g s\n", g_duration);

            recordStartPoint();
    #endif

    gpuErrchk(hipMemset(g_resArr, 0, numOps * sizeof(DataVector)));

    if (coil.useFastMethod)
        calculateFieldFast<<<blocks, NTHREADS>>>(numOps, coil, g_posArr, g_resArr);
    else
        calculateFieldSlow<<<blocks, NTHREADS>>>(numOps, coil, g_posArr, g_resArr);

    gpuErrchk(hipDeviceSynchronize());

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tCalculations:             %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    if(resArr != nullptr)
        gpuErrchk(hipMemcpy(resArr, g_resArr, numOps * sizeof(DataVector), hipMemcpyDeviceToHost));

#if DEBUG_TIMINGS
    g_duration = getIntervalDuration();
    printf("\tWriting to output array:  %.9g s\n\n", g_duration);

    g_duration = getIntervalDuration();
    printf("\tDevice buffer size:       %.3lf MB\n", (6.0 * double(numOps * sizeof(TYPE)) / 1.0e6));
    printf("\tTotal blocks:             %lli\n", blocks);
    printf("\tThreads per calculation:  %i\n", NTHREADS);
    printf("\tPrecision:                %dx%d\n", coil.thicknessIncrements, coil.angularIncrements);
    printf("\tTotal calculations:       %lli\n", numOps);
    printf("\tTotal MegaIncrements:     %.f\n", 1e-6 * double(numOps * coil.thicknessIncrements * coil.angularIncrements));
    printf("\n\tPerformance:              %.1f kPoints/s\n", double(0.001 * numOps / g_duration));
    printf("---------------------------------------------------\n\n");
#endif
}
