#include "hip/hip_runtime.h"
#include "hardware_acceleration.h"

#include "CUDAConstants.h"
#include "Timing.h"
#include "CUDAErrorCheck.h"
#include "CoilData.h"
#include "GPUMemoryManagement.h"

#include <cstdio>


__global__
void calculateGradientSlow(long long numOps, CoilData coil, const DataVector *posArr, DataMatrix *resArr)
{
    unsigned int index = threadIdx.x;
    long long global_index = blockIdx.x * blockDim.x + index;

    if(global_index >= numOps)
        return;

    TYPE x1 = posArr[global_index].x;
    TYPE y1 = posArr[global_index].y;
    TYPE z1 = posArr[global_index].z;

    x1 -= coil.positionVector[0];
    y1 -= coil.positionVector[1];
    z1 -= coil.positionVector[2];

    TYPE x = x1 * coil.invTransformArray[0] + y1 * coil.invTransformArray[1] + z1 * coil.invTransformArray[2];
    TYPE y = x1 * coil.invTransformArray[3] + y1 * coil.invTransformArray[4] + z1 * coil.invTransformArray[5];
    TYPE z = x1 * coil.invTransformArray[6] + y1 * coil.invTransformArray[7] + z1 * coil.invTransformArray[8];

    TYPE zCoord = z;
    TYPE rCoord = sqrt(x * x + y * y);
    TYPE phiCord = atan2(y, x);

    TYPE bufferValueRP = 0.0f;
    TYPE bufferValueRR = 0.0f;
    TYPE bufferValueRZ = 0.0f;
    TYPE bufferValueZZ = 0.0f;

    for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
    {
        TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

        TYPE tempConstA = incrementPositionT * incrementPositionT;
        TYPE tempConstB = rCoord * rCoord;
        TYPE tempConstC = zCoord * zCoord;
        TYPE tempConstD = rCoord * incrementPositionT;
        TYPE tempConstE = incrementPositionT * zCoord;

        TYPE tempConstF = 2.0f * tempConstA + 2.0f * tempConstB - tempConstC;
        TYPE tempConstG = tempConstA + tempConstB + tempConstC;

        for (int incF = 0; incF < coil.angularIncrements; ++incF)
        {
            TYPE cosinePhi = coil.cosPrecomputeArray[incF];
            TYPE tempConstH = coil.constFactor * coil.thicknessWeightArray[incT] * coil.angularWeightArray[incF];

            TYPE tempConstI = tempConstG - 2.0f * tempConstD * cosinePhi;
            TYPE tempConstJ = tempConstI * sqrt(tempConstI);

            TYPE tempConstX = tempConstH / (tempConstJ);
            TYPE tempConstY = tempConstH / (tempConstJ * tempConstI);

            bufferValueRP += tempConstX * tempConstE * cosinePhi / rCoord;
            bufferValueRR += tempConstY * (-3.0f * tempConstE * (rCoord - incrementPositionT * cosinePhi)) * cosinePhi;
            bufferValueZZ += tempConstY * (-3.0f * tempConstE * (incrementPositionT - rCoord * cosinePhi));
            bufferValueRZ += tempConstY *
                    (incrementPositionT * (tempConstF - tempConstD * cosinePhi) * cosinePhi - 3.0f * tempConstA * rCoord);
        }
    }

    TYPE xxGrad, xyGrad, xzGrad, yxGrad, yyGrad, yzGrad, zxGrad, zyGrad, zzGrad;

    if (rCoord / coil.innerRadius > 1e-5)
    {
        TYPE sinPhi = sin(phiCord);
        TYPE cosPhi = cos(phiCord);

        xxGrad = bufferValueRR * cosPhi * cosPhi + bufferValueRP * sinPhi * sinPhi;
        yyGrad = bufferValueRR * sinPhi * sinPhi + bufferValueRP * cosPhi * cosPhi;
        zzGrad = bufferValueZZ;

        xyGrad = 0.5f * sin(2.0f * phiCord) * (bufferValueRR - bufferValueRP);
        xzGrad = bufferValueRZ * cosPhi;
        yzGrad = bufferValueRZ * sinPhi;

        yxGrad = xyGrad;
        zxGrad = xzGrad;
        zyGrad = yzGrad;
    }
    else
    {
        xxGrad = bufferValueRR;
        yyGrad = bufferValueRR;
        zzGrad = bufferValueZZ;

        xyGrad = 0.f;
        xzGrad = 0.f;
        yxGrad = 0.f;
        yzGrad = 0.f;
        zxGrad = 0.f;
        zyGrad = 0.f;
    }

    TYPE xxRes = coil.transformArray[0] * xxGrad + coil.transformArray[1] * yxGrad + coil.transformArray[2] * zxGrad;
    TYPE xyRes = coil.transformArray[0] * xyGrad + coil.transformArray[1] * yyGrad + coil.transformArray[2] * zyGrad;
    TYPE xzRes = coil.transformArray[0] * xzGrad + coil.transformArray[1] * yzGrad + coil.transformArray[2] * zzGrad;
    TYPE yxRes = coil.transformArray[3] * xxGrad + coil.transformArray[4] * yxGrad + coil.transformArray[5] * zxGrad;
    TYPE yyRes = coil.transformArray[3] * xyGrad + coil.transformArray[4] * yyGrad + coil.transformArray[5] * zyGrad;
    TYPE yzRes = coil.transformArray[3] * xzGrad + coil.transformArray[4] * yzGrad + coil.transformArray[5] * zzGrad;
    TYPE zxRes = coil.transformArray[6] * xxGrad + coil.transformArray[7] * yxGrad + coil.transformArray[8] * zxGrad;
    TYPE zyRes = coil.transformArray[6] * xyGrad + coil.transformArray[7] * yyGrad + coil.transformArray[8] * zyGrad;
    TYPE zzRes = coil.transformArray[6] * xzGrad + coil.transformArray[7] * yzGrad + coil.transformArray[8] * zzGrad;

    resArr[global_index].xx = xxRes;
    resArr[global_index].xy = xyRes;
    resArr[global_index].xz = xzRes;
    resArr[global_index].yx = yxRes;
    resArr[global_index].yy = yyRes;
    resArr[global_index].yz = yzRes;
    resArr[global_index].zx = zxRes;
    resArr[global_index].zy = zyRes;
    resArr[global_index].zz = zzRes;
}

__global__
void calculateGradientFast(long long numOps, CoilData coil, const DataVector *posArr, DataMatrix *resArr)
{
    unsigned int index = threadIdx.x;
    long long global_index = blockIdx.x * blockDim.x + index;

    if(global_index >= numOps)
        return;

    TYPE x1 = posArr[global_index].x;
    TYPE y1 = posArr[global_index].y;
    TYPE z1 = posArr[global_index].z;

    x1 -= coil.positionVector[0];
    y1 -= coil.positionVector[1];
    z1 -= coil.positionVector[2];

    TYPE x = x1 * coil.invTransformArray[0] + y1 * coil.invTransformArray[1] + z1 * coil.invTransformArray[2];
    TYPE y = x1 * coil.invTransformArray[3] + y1 * coil.invTransformArray[4] + z1 * coil.invTransformArray[5];
    TYPE z = x1 * coil.invTransformArray[6] + y1 * coil.invTransformArray[7] + z1 * coil.invTransformArray[8];

    TYPE zCoord = z;
    TYPE rCoord = sqrt(x * x + y * y);
    TYPE phiCord = atan2(y, x);

    TYPE bufferValueRP = 0.0f;
    TYPE bufferValueRR = 0.0f;
    TYPE bufferValueRZ = 0.0f;
    TYPE bufferValueZZ = 0.0f;

    TYPE topEdge = zCoord + 0.5f * coil.length;
    TYPE bottomEdge = zCoord - 0.5f * coil.length;

    for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
    {
        TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

        TYPE tempConstA = incrementPositionT * incrementPositionT;
        TYPE tempConstB = rCoord * rCoord;
        TYPE tempConstC = incrementPositionT * rCoord;

        TYPE tempConstD = tempConstA + tempConstB;
        TYPE tempConstE = tempConstA * tempConstA + tempConstB * tempConstB;
        TYPE tempConstF = tempConstC * tempConstC;

        TYPE tempConstG1 = tempConstD + topEdge * topEdge;
        TYPE tempConstG2 = tempConstD + bottomEdge * bottomEdge;

        for (int incF = 0; incF < coil.angularIncrements; ++incF)
        {
            TYPE cosinePhi = coil.cosPrecomputeArray[incF];
            TYPE cosinePhi2 = cosinePhi * cosinePhi;
            TYPE phiExpression = 2.0f * tempConstC * cosinePhi;

            TYPE tempConstI = coil.constFactor * coil.thicknessWeightArray[incT] * coil.angularWeightArray[incF];

            TYPE tempConstJ1 = tempConstG1 - phiExpression;
            TYPE tempConstJ2 = tempConstG2 - phiExpression;

            TYPE tempConstK1 = rsqrt(tempConstJ1);
            TYPE tempConstK2 = rsqrt(tempConstJ2);

            TYPE tempConstL1 = tempConstK1 / (tempConstJ1);
            TYPE tempConstL2 = tempConstK2 / (tempConstJ2);

            TYPE tempConstM = tempConstD - phiExpression;
            TYPE tempConstN =
                    2.0f * tempConstF * cosinePhi * (cosinePhi2 + 2.0f) -
                    tempConstC * (3.0f * cosinePhi2 + 1.0f) * tempConstD + cosinePhi * tempConstE;
            TYPE tempConstO = cosinePhi * tempConstD - 2.0f * tempConstC;

            bufferValueRP += tempConstI * (incrementPositionT * cosinePhi / rCoord) * (tempConstK2 - tempConstK1);
            bufferValueRR += tempConstI * (tempConstC - tempConstA * cosinePhi) * cosinePhi * (tempConstL1 - tempConstL2);
            bufferValueZZ += tempConstI * (tempConstA - tempConstC * cosinePhi) * (tempConstL1 - tempConstL2);
            bufferValueRZ += tempConstI * incrementPositionT / (tempConstM * tempConstM) *
                             (topEdge * tempConstL1 * (tempConstO * tempConstJ1 + tempConstN) -
                              bottomEdge * tempConstL2 * (tempConstO * tempConstJ2 + tempConstN));
        }
    }

    TYPE xxGrad, xyGrad, xzGrad, yxGrad, yyGrad, yzGrad, zxGrad, zyGrad, zzGrad;

    if (rCoord / coil.innerRadius > 1e-5)
    {
        TYPE sinPhi = sin(phiCord);
        TYPE cosPhi = cos(phiCord);

        xxGrad = bufferValueRR * cosPhi * cosPhi + bufferValueRP * sinPhi * sinPhi;
        yyGrad = bufferValueRR * sinPhi * sinPhi + bufferValueRP * cosPhi * cosPhi;
        zzGrad = bufferValueZZ;

        xyGrad = 0.5f * sin(2.0f * phiCord) * (bufferValueRR - bufferValueRP);
        xzGrad = bufferValueRZ * cosPhi;
        yzGrad = bufferValueRZ * sinPhi;

        yxGrad = xyGrad;
        zxGrad = xzGrad;
        zyGrad = yzGrad;
    }
    else
    {
        xxGrad = bufferValueRR;
        yyGrad = bufferValueRR;
        zzGrad = bufferValueZZ;

        xyGrad = 0.f;
        xzGrad = 0.f;
        yxGrad = 0.f;
        yzGrad = 0.f;
        zxGrad = 0.f;
        zyGrad = 0.f;
    }

    TYPE xxRes = coil.transformArray[0] * xxGrad + coil.transformArray[1] * yxGrad + coil.transformArray[2] * zxGrad;
    TYPE xyRes = coil.transformArray[0] * xyGrad + coil.transformArray[1] * yyGrad + coil.transformArray[2] * zyGrad;
    TYPE xzRes = coil.transformArray[0] * xzGrad + coil.transformArray[1] * yzGrad + coil.transformArray[2] * zzGrad;
    TYPE yxRes = coil.transformArray[3] * xxGrad + coil.transformArray[4] * yxGrad + coil.transformArray[5] * zxGrad;
    TYPE yyRes = coil.transformArray[3] * xyGrad + coil.transformArray[4] * yyGrad + coil.transformArray[5] * zyGrad;
    TYPE yzRes = coil.transformArray[3] * xzGrad + coil.transformArray[4] * yzGrad + coil.transformArray[5] * zzGrad;
    TYPE zxRes = coil.transformArray[6] * xxGrad + coil.transformArray[7] * yxGrad + coil.transformArray[8] * zxGrad;
    TYPE zyRes = coil.transformArray[6] * xyGrad + coil.transformArray[7] * yyGrad + coil.transformArray[8] * zyGrad;
    TYPE zzRes = coil.transformArray[6] * xzGrad + coil.transformArray[7] * yzGrad + coil.transformArray[8] * zzGrad;

    resArr[global_index].xx = xxRes;
    resArr[global_index].xy = xyRes;
    resArr[global_index].xz = xzRes;
    resArr[global_index].yx = yxRes;
    resArr[global_index].yy = yyRes;
    resArr[global_index].yz = yzRes;
    resArr[global_index].zx = zxRes;
    resArr[global_index].zy = zyRes;
    resArr[global_index].zz = zzRes;
}
	
namespace 
{
    DataVector *g_posArr = nullptr;
    DataMatrix *g_resArr = nullptr;

    void getBuffers(long long numOps)
    {
        std::vector<void*> buffers = GPUMem::getBuffers(
            {numOps * (long long)sizeof(DataVector), numOps * (long long)sizeof(DataMatrix)}
        );

        g_posArr = static_cast<DataVector*>(buffers[0]);
        g_resArr = static_cast<DataMatrix*>(buffers[1]);
    }
    
    #if DEBUG_TIMINGS
        double g_duration;
    #endif
}


void Calculate_hardware_accelerated_g(long long numOps, CoilData coil, const DataVector *posArr, DataMatrix *resArr)
{
    #if DEBUG_TIMINGS
        recordStartPoint();
        recordStartPoint();
    #endif

    long long blocks = ceil(double(numOps) / NTHREADS);

    getBuffers(numOps);

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tResource startup:         %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    gpuErrchk(hipMemcpy(g_posArr, posArr, numOps * sizeof(DataVector), hipMemcpyHostToDevice));

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tMemory initialization:    %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    if (coil.useFastMethod)
        calculateGradientFast<<<blocks, NTHREADS>>>(numOps, coil, g_posArr, g_resArr);
    else
        calculateGradientSlow<<<blocks, NTHREADS>>>(numOps, coil, g_posArr, g_resArr);

	gpuErrchk(hipDeviceSynchronize());

	#if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tCalculations:             %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    if(resArr != nullptr)
        gpuErrchk(hipMemcpy(resArr, g_resArr, numOps * sizeof(DataMatrix), hipMemcpyDeviceToHost));

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tWriting to output array:  %.9g s\n\n", g_duration);

        g_duration = getIntervalDuration();
        printf("\tDevice buffer size:       %.3lf MB\n", (12.0 * double(numOps * sizeof(TYPE)) / 1.0e6));
        printf("\tTotal blocks:             %lli\n", blocks);
        printf("\tThreads per calculation:  %i\n", NTHREADS);
        printf("\tPrecision:                %dx%d\n", coil.thicknessIncrements, coil.angularIncrements);
        printf("\tTotal calculations:       %lli\n", numOps);
        printf("\tTotal MegaIncrements:     %.f\n", 1e-6 * double(numOps * coil.thicknessIncrements * coil.angularIncrements));
        printf("\n\tPerformance:              %.1f kPoints/s\n", double(0.001 * numOps / g_duration));
        printf("---------------------------------------------------\n\n");
    #endif
}
