#include "hip/hip_runtime.h"
#include "CoilGroupAcceleration.h"

#include "Timing.h"
#include "CUDAUtils/ErrorCheck/CUDAErrorCheck.h"
#include "CUDAUtils/MemoryManagement/GPUMemoryManagement.h"

#include <cstdio>


__global__
void calculatePotentialGroup(long long opCount, long long coilIndex,
                             const CoilData *coilArr,
                             const VectorData *posArr,
                             VectorData *resArr)
{
    unsigned int index = threadIdx.x;
    long long globalIndex = (long long) blockIdx.x * blockDim.x + index;

    if(globalIndex >= opCount)
        return;

    __shared__ CoilData coil;
    coil = coilArr[coilIndex];

    TYPE x1 = posArr[globalIndex].x;
    TYPE y1 = posArr[globalIndex].y;
    TYPE z1 = posArr[globalIndex].z;

    x1 -= coil.positionVector[0];
    y1 -= coil.positionVector[1];
    z1 -= coil.positionVector[2];

    TYPE x = x1 * coil.invTransformArray[0] + y1 * coil.invTransformArray[1] + z1 * coil.invTransformArray[2];
    TYPE y = x1 * coil.invTransformArray[3] + y1 * coil.invTransformArray[4] + z1 * coil.invTransformArray[5];
    TYPE z = x1 * coil.invTransformArray[6] + y1 * coil.invTransformArray[7] + z1 * coil.invTransformArray[8];

    TYPE zCoord = z;
    TYPE rCoord = sqrt(x * x + y * y);
    TYPE phiCord = atan2(y, x);

    TYPE potential = 0.0f;

    TYPE topEdge = zCoord + 0.5f * coil.length;
    TYPE bottomEdge = zCoord - 0.5f * coil.length;

    int angIncs = coil.angularIncrements / 2 + coil.angularIncrements % 2;

    if (coil.useFastMethod)
    {
        for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
        {
            TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

            TYPE tempConstA = incrementPositionT * incrementPositionT + rCoord * rCoord;
            TYPE tempConstB = 2.0f * incrementPositionT * rCoord;
            TYPE tempConstG = coil.thicknessWeightArray[incT] * incrementPositionT;

//            for (int incF = 0; incF < coil.angularIncrements; ++incF)
//            {
//                TYPE cosinePhi = coil.cosPrecomputeArray[incF];
//
//                TYPE tempConstC = rsqrt(tempConstA - tempConstB * cosinePhi);
//
//                TYPE tempConstD1 = topEdge * tempConstC;
//                TYPE tempConstD2 = bottomEdge * tempConstC;
//
//                TYPE tempConstE1 = sqrt(tempConstD1 * tempConstD1 + 1.0f);
//                TYPE tempConstE2 = sqrt(tempConstD2 * tempConstD2 + 1.0f);
//
//                TYPE tempConstF = log((tempConstE1 + tempConstD1) / (tempConstE2 + tempConstD2));
//
//                potential += tempConstG * coil.angularWeightArray[incF] * cosinePhi * tempConstF;
//            }

            for (int incF = 0; incF < angIncs; ++incF)
            {
                TYPE cosinePhi_0 = coil.cosPrecomputeArray[2 * incF + 0];
                TYPE cosinePhi_1 = coil.cosPrecomputeArray[2 * incF + 1];

                TYPE tempConstC_0 = rsqrt(tempConstA - tempConstB * cosinePhi_0);
                TYPE tempConstC_1 = rsqrt(tempConstA - tempConstB * cosinePhi_1);

                TYPE tempConstD1_0 = topEdge * tempConstC_0;
                TYPE tempConstD1_1 = topEdge * tempConstC_1;

                TYPE tempConstD2_0 = bottomEdge * tempConstC_0;
                TYPE tempConstD2_1 = bottomEdge * tempConstC_1;

                TYPE tempConstE1_0 = sqrt(tempConstD1_0 * tempConstD1_0 + 1.0f);
                TYPE tempConstE1_1 = sqrt(tempConstD1_1 * tempConstD1_1 + 1.0f);

                TYPE tempConstE2_0 = sqrt(tempConstD2_0 * tempConstD2_0 + 1.0f);
                TYPE tempConstE2_1 = sqrt(tempConstD2_1 * tempConstD2_1 + 1.0f);

                TYPE tempConstF_0 = log((tempConstE1_0 + tempConstD1_0) / (tempConstE2_0 + tempConstD2_0));
                TYPE tempConstF_1 = log((tempConstE1_1 + tempConstD1_1) / (tempConstE2_1 + tempConstD2_1));

                potential += tempConstG * coil.angularWeightArray[2 * incF + 0] * cosinePhi_0 * tempConstF_0;
                potential += tempConstG * coil.angularWeightArray[2 * incF + 1] * cosinePhi_1 * tempConstF_1;
            }
        }
    }
    else
    {
        for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
        {
            TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

            TYPE tempConstA = incrementPositionT * incrementPositionT + rCoord * rCoord + zCoord * zCoord;
            TYPE tempConstB = 2.0f * incrementPositionT * rCoord;
            TYPE tempConstC = coil.thicknessWeightArray[incT] * incrementPositionT;

//            for (int incF = 0; incF < coil.angularIncrements; ++incF)
//            {
//                TYPE cosinePhi = coil.cosPrecomputeArray[incF];
//
//                TYPE tempConstD = rsqrt(tempConstA - tempConstB * cosinePhi);
//
//                potential += tempConstC * coil.angularWeightArray[incF] * cosinePhi * tempConstD;
//            }

            for (int incF = 0; incF < angIncs; ++incF)
            {
                TYPE cosinePhi_0 = coil.cosPrecomputeArray[2 * incF + 0];
                TYPE cosinePhi_1 = coil.cosPrecomputeArray[2 * incF + 1];

                TYPE tempConstD_0 = rsqrt(tempConstA - tempConstB * cosinePhi_0);
                TYPE tempConstD_1 = rsqrt(tempConstA - tempConstB * cosinePhi_1);

                potential += tempConstC * coil.angularWeightArray[2 * incF + 0] * cosinePhi_0 * tempConstD_0;
                potential += tempConstC * coil.angularWeightArray[2 * incF + 1] * cosinePhi_1 * tempConstD_1;

            }
        }
    }

    potential *= coil.constFactor;

    TYPE xPot = (-1.0f) * sin(phiCord) * potential;
    TYPE yPot = potential * cos(phiCord);
    TYPE zPot = 0.0f;

    TYPE xRes = xPot * coil.transformArray[0] + yPot * coil.transformArray[1] + zPot * coil.transformArray[2];
    TYPE yRes = xPot * coil.transformArray[3] + yPot * coil.transformArray[4] + zPot * coil.transformArray[5];
    TYPE zRes = xPot * coil.transformArray[6] + yPot * coil.transformArray[7] + zPot * coil.transformArray[8];

    resArr[globalIndex].x += xRes;
    resArr[globalIndex].y += yRes;
    resArr[globalIndex].z += zRes;
}


namespace
{
    CoilData *g_coilArr = nullptr;
    VectorData *g_posArr = nullptr;
    VectorData *g_resArr = nullptr;

    void getBuffers(long long coilCount, long long opCount)
    {
        std::vector<void*> buffers = GPUMem::getBuffers(
                { coilCount * (long long)sizeof(CoilData),
                  opCount * (long long)sizeof(VectorData),
                  opCount * (long long)sizeof(VectorData)}
        );

        g_coilArr = static_cast<CoilData*>(buffers[0]);
        g_posArr = static_cast<VectorData*>(buffers[1]);
        g_resArr = static_cast<VectorData*>(buffers[2]);
    }

    #if DEBUG_TIMINGS
        double g_duration;
    #endif
}


void Calculate_hardware_accelerated_a_group(long long coilCount, long long opCount,
                                            const CoilData *coilArr,
                                            const VectorData *posArr,
                                            VectorData *resArr)
{
    #if DEBUG_TIMINGS
        recordStartPoint();
        recordStartPoint();
    #endif

    int blocks = int(std::ceil(double(opCount) / NTHREADS));

    getBuffers(coilCount, opCount);

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tResource startup:         %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    gpuErrchk(hipMemcpy(g_coilArr, coilArr, coilCount * sizeof(CoilData), hipMemcpyHostToDevice))
    gpuErrchk(hipMemcpy(g_posArr, posArr, opCount * sizeof(VectorData), hipMemcpyHostToDevice))

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tMemory initialization:    %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    gpuErrchk(hipMemset(g_resArr, 0, opCount * sizeof(VectorData)))

    for (int i = 0; i < coilCount; ++i)
    {
        calculatePotentialGroup<<<blocks, NTHREADS>>>(
            opCount, i, g_coilArr, g_posArr, g_resArr
        );
        gpuErrchk(hipDeviceSynchronize())
    }

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tCalculations:             %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    if(resArr != nullptr)
        gpuErrchk(hipMemcpy(resArr, g_resArr, opCount * sizeof(VectorData), hipMemcpyDeviceToHost))

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tWriting to output array:  %.9g s\n\n", g_duration);

        g_duration = getIntervalDuration();
        printf("\tDevice buffer size:       %.3lf MB\n", (6.0 * double(opCount * sizeof(TYPE) + coilCount * sizeof(CoilData)) / 1.0e6));
        printf("\tTotal blocks:             %lli\n", blocks);
        printf("\tThreads per calculation:  %i\n", NTHREADS);
        printf("\tTotal coils:              %lli\n", coilCount);
        printf("\tTotal points:             %lli\n", opCount);
        printf("\tTotal calculations:       %lli\n", opCount * coilCount);
        printf("\n\tPerformance:              %.1f kPoints/s\n", double(0.001 * opCount / g_duration));
        printf("\n\tEffectivePerformance:     %.1f kPoints/s\n", double(0.001 * opCount * coilCount / g_duration));
        printf("---------------------------------------------------\n\n");
    #endif
}
