#include "hip/hip_runtime.h"
#include "CoilGroupAcceleration.h"

#include "Timing.h"
#include "CUDAUtils/ErrorCheck/CUDAErrorCheck.h"
#include "CUDAUtils/MemoryManagement/GPUMemoryManagement.h"

#include <cstdio>


__global__
void calculateFieldEGroup(long long opCount, long long coilIndex,
                          const CoilData *coilArr,
                          const DataVector *posArr,
                          DataVector *resArr)
{
    unsigned int index = threadIdx.x;
    long long globalIndex = blockIdx.x * blockDim.x + index;

    if(globalIndex >= opCount)
        return;

    __shared__ CoilData coil;
    coil = coilArr[coilIndex];

    TYPE x1 = posArr[globalIndex].x;
    TYPE y1 = posArr[globalIndex].y;
    TYPE z1 = posArr[globalIndex].z;

    x1 -= coil.positionVector[0];
    y1 -= coil.positionVector[1];
    z1 -= coil.positionVector[2];

    TYPE x = x1 * coil.invTransformArray[0] + y1 * coil.invTransformArray[1] + z1 * coil.invTransformArray[2];
    TYPE y = x1 * coil.invTransformArray[3] + y1 * coil.invTransformArray[4] + z1 * coil.invTransformArray[5];
    TYPE z = x1 * coil.invTransformArray[6] + y1 * coil.invTransformArray[7] + z1 * coil.invTransformArray[8];

    TYPE zCoord = z;
    TYPE rCoord = sqrt(x * x + y * y);
    TYPE phiCord = atan2(y, x);

    TYPE potential = 0.0f;

    TYPE topEdge = zCoord + 0.5f * coil.length;
    TYPE bottomEdge = zCoord - 0.5f * coil.length;

    if (coil.useFastMethod)
    {
        for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
        {
            TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

            TYPE tempConstA = incrementPositionT * incrementPositionT + rCoord * rCoord;
            TYPE tempConstB = 2.0f * incrementPositionT * rCoord;

            for (int incF = 0; incF < coil.angularIncrements; ++incF)
            {
                TYPE cosinePhi = coil.cosPrecomputeArray[incF];

                TYPE tempConstC = rsqrt(tempConstA - tempConstB * cosinePhi);

                TYPE tempConstD1 = topEdge * tempConstC;
                TYPE tempConstD2 = bottomEdge * tempConstC;

                TYPE tempConstE1 = sqrt(tempConstD1 * tempConstD1 + 1.0f);
                TYPE tempConstE2 = sqrt(tempConstD2 * tempConstD2 + 1.0f);

                TYPE tempConstF = log((tempConstE1 + tempConstD1) / (tempConstE2 + tempConstD2));

                potential += coil.constFactor *
                             coil.thicknessWeightArray[incT] * coil.angularWeightArray[incF] *
                             incrementPositionT * cosinePhi * tempConstF;
            }
        }
    }
    else
    {
        for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
        {
            TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

            TYPE tempConstA = incrementPositionT * incrementPositionT + rCoord * rCoord + zCoord * zCoord;
            TYPE tempConstB = 2.0f * incrementPositionT * rCoord;

            for (int incF = 0; incF < coil.angularIncrements; ++incF)
            {
                TYPE cosinePhi = coil.cosPrecomputeArray[incF];

                TYPE tempConstC = rsqrt(tempConstA - tempConstB * cosinePhi);

                potential += coil.constFactor *
                             coil.thicknessWeightArray[incT] * coil.angularWeightArray[incF] *
                             incrementPositionT * cosinePhi * tempConstC;
            }
        }
    }

    TYPE xPot = (-1.0f) * sin(phiCord) * potential;
    TYPE yPot = potential * cos(phiCord);
    TYPE zPot = 0.0f;

    TYPE xRes = xPot * coil.transformArray[0] + yPot * coil.transformArray[1] + zPot * coil.transformArray[2];
    TYPE yRes = xPot * coil.transformArray[3] + yPot * coil.transformArray[4] + zPot * coil.transformArray[5];
    TYPE zRes = xPot * coil.transformArray[6] + yPot * coil.transformArray[7] + zPot * coil.transformArray[8];

    resArr[globalIndex].x += xRes;
    resArr[globalIndex].y += yRes;
    resArr[globalIndex].z += zRes;
}


namespace
{
    CoilData *g_coilArr = nullptr;
    DataVector *g_posArr = nullptr;
    DataVector *g_resArr = nullptr;

    void getBuffers(long long coilCount, long long opCount)
    {
        std::vector<void*> buffers = GPUMem::getBuffers(
                { coilCount * (long long)sizeof(CoilData),
                  opCount * (long long)sizeof(DataVector),
                  opCount * (long long)sizeof(DataVector)}
        );

        g_coilArr = static_cast<CoilData*>(buffers[0]);
        g_posArr = static_cast<DataVector*>(buffers[1]);
        g_resArr = static_cast<DataVector*>(buffers[2]);
    }

    #if DEBUG_TIMINGS
        double g_duration;
    #endif
}


void Calculate_hardware_accelerated_a_group(long long coilCount, long long opCount,
                                            const CoilData *coilArr,
                                            const DataVector *posArr,
                                            DataVector *resArr)
{
    #if DEBUG_TIMINGS
        recordStartPoint();
        recordStartPoint();
    #endif

    long long blocks = ceil(double(opCount) / NTHREADS);

    getBuffers(coilCount, opCount);

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tResource startup:         %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    gpuErrchk(hipMemcpy(g_coilArr, coilArr, coilCount * sizeof(CoilData), hipMemcpyHostToDevice))
    gpuErrchk(hipMemcpy(g_posArr, posArr, opCount * sizeof(DataVector), hipMemcpyHostToDevice))

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tMemory initialization:    %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    gpuErrchk(hipMemset(g_resArr, 0, opCount * sizeof(DataVector)))

    for (int i = 0; i < coilCount; ++i)
    {
        calculateFieldEGroup<<<blocks, NTHREADS>>>(
            opCount, i, g_coilArr, g_posArr, g_resArr
        );
        gpuErrchk(hipDeviceSynchronize())
    }

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tCalculations:             %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    if(resArr != nullptr)
        gpuErrchk(hipMemcpy(resArr, g_resArr, opCount * sizeof(DataVector), hipMemcpyDeviceToHost))

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tWriting to output array:  %.9g s\n\n", g_duration);

        g_duration = getIntervalDuration();
        printf("\tDevice buffer size:       %.3lf MB\n", (6.0 * double(opCount * sizeof(TYPE) + coilCount * sizeof(CoilData)) / 1.0e6));
        printf("\tTotal blocks:             %lli\n", blocks);
        printf("\tThreads per calculation:  %i\n", NTHREADS);
        printf("\tTotal coils:              %lli\n", coilCount);
        printf("\tTotal points:             %lli\n", opCount);
        printf("\tTotal calculations:       %lli\n", opCount * coilCount);
        printf("\n\tPerformance:              %.1f kPoints/s\n", double(0.001 * opCount / g_duration));
        printf("\n\tEffectivePerformance:     %.1f kPoints/s\n", double(0.001 * opCount * coilCount / g_duration));
        printf("---------------------------------------------------\n\n");
    #endif
}
