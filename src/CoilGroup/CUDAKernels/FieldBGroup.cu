#include "hip/hip_runtime.h"
#include "CoilGroupAcceleration.h"

#include "Timing.h"
#include "CUDAUtils/ErrorCheck/CUDAErrorCheck.h"
#include "CUDAUtils/MemoryManagement/GPUMemoryManagement.h"

#include <cstdio>


__global__
void calculateFieldGroup(long long opCount, long long coilIndex,
                         const CoilData *coilArr,
                         const VectorData *posArr,
                         VectorData *resArr)
{
    unsigned int index = threadIdx.x;
    long long globalIndex = blockIdx.x * blockDim.x + index;

    if(globalIndex >= opCount)
        return;

    __shared__ CoilData coil;
    coil = coilArr[coilIndex];

    TYPE x1 = posArr[globalIndex].x;
    TYPE y1 = posArr[globalIndex].y;
    TYPE z1 = posArr[globalIndex].z;

    x1 -= coil.positionVector[0];
    y1 -= coil.positionVector[1];
    z1 -= coil.positionVector[2];

    TYPE x = x1 * coil.invTransformArray[0] + y1 * coil.invTransformArray[1] + z1 * coil.invTransformArray[2];
    TYPE y = x1 * coil.invTransformArray[3] + y1 * coil.invTransformArray[4] + z1 * coil.invTransformArray[5];
    TYPE z = x1 * coil.invTransformArray[6] + y1 * coil.invTransformArray[7] + z1 * coil.invTransformArray[8];

    TYPE zCoord = z;
    TYPE rCoord = sqrt(x * x + y * y);
    TYPE phiCord = atan2(y, x);

    TYPE fieldH = 0.0f;
    TYPE fieldZ = 0.0f;

    TYPE topEdge = zCoord + 0.5f * coil.length;
    TYPE bottomEdge = zCoord - 0.5f * coil.length;

    if (coil.useFastMethod)
    {
        for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
        {
            TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

            TYPE tempConstA = incrementPositionT * incrementPositionT;
            TYPE tempConstB = 2.0f * incrementPositionT * rCoord;
            TYPE tempConstC = tempConstA + rCoord * rCoord;

            TYPE tempConstD1 = topEdge * topEdge + tempConstC;
            TYPE tempConstD2 = bottomEdge * bottomEdge + tempConstC;

            for (int incF = 0; incF < coil.angularIncrements; ++incF)
            {
                TYPE cosinePhi = coil.cosPrecomputeArray[incF];

                TYPE tempConstE = tempConstB * cosinePhi;

                TYPE tempConstF1 = rsqrt(tempConstD1 - tempConstE);
                TYPE tempConstF2 = rsqrt(tempConstD2 - tempConstE);

                TYPE tempConstG = coil.constFactor * coil.thicknessWeightArray[incT] * coil.angularWeightArray[incF];

                fieldH += tempConstG * incrementPositionT * cosinePhi * (tempConstF2 - tempConstF1);
                fieldZ += tempConstG *
                          ((tempConstA - 0.5f * tempConstE) / (tempConstC - tempConstE)) *
                          (topEdge * tempConstF1 - bottomEdge * tempConstF2);
            }
        }
    }
    else
    {
        for (int incT = 0; incT < coil.thicknessIncrements; ++incT)
        {
            TYPE incrementPositionT = coil.innerRadius + 0.5f * coil.thickness * (1.0f + coil.thicknessPositionArray[incT]);

            TYPE tempConstA = incrementPositionT * incrementPositionT;
            TYPE tempConstB = incrementPositionT * rCoord;
            TYPE tempConstC = tempConstA + rCoord * rCoord + zCoord * zCoord;
            TYPE tempConstD = incrementPositionT * zCoord;

            for (int incF = 0; incF < coil.angularIncrements; ++incF)
            {
                TYPE cosinePhi = coil.cosPrecomputeArray[incF];

                TYPE tempConstE = tempConstC - 2.0f * tempConstB * cosinePhi;
                TYPE tempConstF = tempConstE * sqrt(tempConstE);
                TYPE tempConstG = coil.constFactor * coil.thicknessWeightArray[incT] * coil.angularWeightArray[incF] / tempConstF;

                fieldH += tempConstG * (tempConstD * cosinePhi);
                fieldZ += tempConstG * (tempConstA - tempConstB * cosinePhi);
            }
        }
    }

    TYPE xField = fieldH * cos(phiCord);
    TYPE yField = fieldH * sin(phiCord);
    TYPE zField = fieldZ;

    TYPE xRes = xField * coil.transformArray[0] + yField * coil.transformArray[1] + zField * coil.transformArray[2];
    TYPE yRes = xField * coil.transformArray[3] + yField * coil.transformArray[4] + zField * coil.transformArray[5];
    TYPE zRes = xField * coil.transformArray[6] + yField * coil.transformArray[7] + zField * coil.transformArray[8];

    resArr[globalIndex].x += xRes;
    resArr[globalIndex].y += yRes;
    resArr[globalIndex].z += zRes;
}


namespace
{
    CoilData *g_coilArr = nullptr;
    VectorData *g_posArr = nullptr;
    VectorData *g_resArr = nullptr;

    void getBuffers(long long coilCount, long long opCount)
    {
        std::vector<void*> buffers = GPUMem::getBuffers(
                { coilCount * (long long)sizeof(CoilData),
                  opCount * (long long)sizeof(VectorData),
                  opCount * (long long)sizeof(VectorData)}
        );

        g_coilArr = static_cast<CoilData*>(buffers[0]);
        g_posArr = static_cast<VectorData*>(buffers[1]);
        g_resArr = static_cast<VectorData*>(buffers[2]);
    }

    #if DEBUG_TIMINGS
        double g_duration;
    #endif
}


void Calculate_hardware_accelerated_b_group(long long coilCount, long long opCount,
                                            const CoilData *coilArr,
                                            const VectorData *posArr,
                                            VectorData *resArr)
{
    #if DEBUG_TIMINGS
        recordStartPoint();
        recordStartPoint();
    #endif

    long long blocks = ceil(double(opCount) / NTHREADS);

    getBuffers(coilCount, opCount);

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tResource startup:         %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    gpuErrchk(hipMemcpy(g_coilArr, coilArr, coilCount * sizeof(CoilData), hipMemcpyHostToDevice))
    gpuErrchk(hipMemcpy(g_posArr, posArr, opCount * sizeof(VectorData), hipMemcpyHostToDevice))

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tMemory initialization:    %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    gpuErrchk(hipMemset(g_resArr, 0, opCount * sizeof(VectorData)))

    for (int i = 0; i < coilCount; ++i)
    {
        calculateFieldGroup<<<blocks, NTHREADS>>>(
            opCount, i, g_coilArr, g_posArr, g_resArr
        );
        gpuErrchk(hipDeviceSynchronize())
    }

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tCalculations:             %.9g s\n", g_duration);

        recordStartPoint();
    #endif

    if(resArr != nullptr)
        gpuErrchk(hipMemcpy(resArr, g_resArr, opCount * sizeof(VectorData), hipMemcpyDeviceToHost))

    #if DEBUG_TIMINGS
        g_duration = getIntervalDuration();
        printf("\tWriting to output array:  %.9g s\n\n", g_duration);

        g_duration = getIntervalDuration();
        printf("\tDevice buffer size:       %.3lf MB\n", (6.0 * double(opCount * sizeof(TYPE) + coilCount * sizeof(CoilData)) / 1.0e6));
        printf("\tTotal blocks:             %lli\n", blocks);
        printf("\tThreads per calculation:  %i\n", NTHREADS);
        printf("\tTotal coils:              %lli\n", coilCount);
        printf("\tTotal points:             %lli\n", opCount);
        printf("\tTotal calculations:       %lli\n", opCount * coilCount);
        printf("\n\tPerformance:              %.1f kPoints/s\n", double(0.001 * opCount / g_duration));
        printf("\n\tEffectivePerformance:     %.1f kPoints/s\n", double(0.001 * opCount * coilCount / g_duration));
        printf("---------------------------------------------------\n\n");
    #endif
}
